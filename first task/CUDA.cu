#include <iostream>
#include "hip/hip_runtime.h"


hipError_t printWithCuda(unsigned int size);
__global__ void messageKernel(unsigned int size);

int main() {
    const unsigned int size = 5;
    hipError_t cudaStatus;

    if ((cudaStatus = printWithCuda(size)) != hipSuccess) {
        std::cerr << "CUDA operation failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    if ((cudaStatus = hipDeviceReset()) != hipSuccess) {
        std::cerr << "Device reset failed: " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    return 0;
}

__global__ void messageKernel(unsigned int size) {
    const unsigned int tid = threadIdx.x;
    if (tid < size) {
        printf("Message from kernel thread %u\n", tid);
    }
}

hipError_t printWithCuda(unsigned int size) {
    hipError_t status;

    if ((status = hipSetDevice(0)) != hipSuccess) {
        std::cerr << "Device selection error: " << hipGetErrorString(status) << std::endl;
        return status;
    }

    const dim3 blockDims(size);
    const dim3 gridDims(1);
    
    messageKernel<<<gridDims, blockDims>>>(size);
    
    if ((status = hipGetLastError()) != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(status) << std::endl;
        return status;
    }

    if ((status = hipDeviceSynchronize()) != hipSuccess) {
        std::cerr << "Device sync error: " << hipGetErrorString(status) << std::endl;
    }

    return status;
}
